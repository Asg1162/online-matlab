#pragma once
#include <hip/hip_runtime.h>


__global__ void
OmgSin_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = sinf(bufferin[gid]);
}


__global__ void
OmgCos_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = cosf(bufferin[gid]);
}


__global__ void
OmgTan_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = tanf(bufferin[gid]);
}


__global__ void
OmgCot_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = 1.0f/tanf(bufferin[gid]);
}


__global__ void
OmgLog_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = logf(bufferin[gid]);
}

__global__ void
OmgLog2_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = log2f(bufferin[gid]);
}

__global__ void
OmgLog10_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = log10f(bufferin[gid]);
}

__global__ void
OmgExp_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = expf(bufferin[gid]);
}

__global__ void
OmgAbs_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = fabsf(bufferin[gid]);
}

__global__ void
OmgFix_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = truncf(bufferin[gid]);
}

__global__ void
OmgCeil_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = ceilf(bufferin[gid]);
}

__global__ void
OmgFloor_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = floorf(bufferin[gid]);
}

__global__ void
OmgRound_kernel(int numElement, const float *bufferin, float *bufferout)
{
  // TODO make it u24
  int blockId = blockIdx.y * gridDim.x +  blockIdx.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int step = gridDim.x * blockDim.x;

    //blockIdx.y * step + blockIdx.x * gridDim.x * blockDim.x;
  for (int gid = tid; gid < numElement; gid += step)
    if (gid < numElement)
      bufferout[gid] = rintf(bufferin[gid]);
}
